#include "algebra.h"
#include <iostream>
#include <cstdlib>
#include <limits>
#include <ctime>
#include <stdio.h>

#define TEST 3

using namespace std;

#if TEST == 1 || TEST == 2

#define BLOCK_SIZE 16
#define N 10

#elif TEST == 3

#define BLOCK_SIZE 32
#define N 4000

#endif


typedef Vector<N, PHOST> Vec;
typedef Vector<N, PDEVICE> d_Vec;
typedef Matrix<N, N, PHOST> Mat;
typedef Matrix<N, N, PDEVICE> d_Mat;

dim3 grid_size(dim3 dimBlock, size_t x, size_t y) {
    dim3 dimGrid(x / dimBlock.x, y / dimBlock.y);
    if(x % dimBlock.x != 0) dimGrid.x++;
    if(y % dimBlock.y != 0) dimGrid.y++;
    return dimGrid;
}

void init_vector(Vec& v) {
    for(int i = 0; i < v.get_width(); i++)
        v.get(i) = i;
}

void init_matrix(Mat& M) {
    for(int i = 0; i < M.get_width(); i++) {
        for(int j = 0; j < M.get_height(); j++) {
            M.get(i, j) = i + j*N;
        }
    }
}

float random_float() {
    return (float)rand() * (float)( rand() % 100000) / (float)rand();
}

void random_matrix(Mat& M) {
    for(int i = 0; i < M.get_width(); i++) {
        for(int j = 0; j < M.get_height(); j++) {
            M.get(i, j) = random_float();
        }
    }
}


int main() {
#if TEST == 1
    Vec v, res;
    Mat A;
    d_Vec d_v, d_res;
    d_Mat d_A;

    A.alloc();
    v.alloc();
    res.alloc();
    d_v.alloc();
    d_A.alloc();
    d_res.alloc();

    init_vector(v);
    init_matrix(A);

    //VecMatMul(v, A, res);

    hipMemcpy(d_A.elements, A.elements, A.get_size(), hipMemcpyHostToDevice);
    hipMemcpy(d_v.elements, v.elements, v.get_size(), hipMemcpyHostToDevice);
    
    dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid = grid_size(dimBlock, res.get_width(), 1);
    VecMatMulKernel<<<dimGrid, dimBlock>>>(d_v, d_A, d_res);

    hipMemcpy(res.elements, d_res.elements, res.get_size(), hipMemcpyDeviceToHost);

    cout << v << endl << A << endl << res << endl;

    A.dealloc();
    v.dealloc();
    res.dealloc();
    d_v.dealloc();
    d_A.dealloc();
    d_res.dealloc();

#elif TEST == 2
    Mat A, D;
    d_Mat d_A, d_D;

    A.alloc();
    D.alloc();
    d_A.alloc();
    d_D.alloc();

    init_matrix(A);

    hipMemcpy(d_A.elements, A.elements, A.get_size(), hipMemcpyHostToDevice);
    
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid = grid_size(dimBlock, A.get_width(), A.get_height());
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_A, d_D);
    
    hipMemcpy(D.elements, d_D.elements, D.get_size(), hipMemcpyDeviceToHost);

    cout << A << endl << D << endl;

    A.dealloc();
    D.dealloc();
    d_A.dealloc();
    d_D.dealloc();

#elif TEST == 3
    Mat A, B, C;
    d_Mat d_A, d_B, d_C;
    srand(time(0));

    A.alloc();
    B.alloc();
    C.alloc();
    d_A.alloc();
    d_B.alloc();
    d_C.alloc();

    random_matrix(A);
    random_matrix(B);

    hipMemcpy(d_A.elements, A.elements, A.get_size(), hipMemcpyHostToDevice);
    hipMemcpy(d_B.elements, B.elements, B.get_size(), hipMemcpyHostToDevice);


    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid = grid_size(dimBlock, N, N);

    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    hipMemcpy(C.elements, d_C.elements, C.get_size(), hipMemcpyDeviceToHost);

    //MatMul(A, B, C);

    if(N < 5)
        cout << A << endl << B << endl << C << endl;

    A.dealloc();
    B.dealloc();
    C.dealloc();
    d_A.dealloc();
    d_B.dealloc();
    d_C.dealloc();
#endif
}
